#include <string>
#include "kernel.cu"

void gpu_blur (std::string filename)
{
    BMP Background;
    Background.ReadFromFile(filename.c_str());
    int height = Background.TellHeight();
    int width = Background.TellWidth();
    int depth = Background.TellBitDepth();
    BMP Output=Background;

    hipError_t cuda_ret;

    int *weights_d;

    int weights[18];


//SET WEIGHTS:
    for(int i = 0; i < 9; i++){
        weights[i] = 1;
    }
    weights[9] = 16;
    weights[10] = 8;
    weights[11] = 16;
    weights[12] = 8;
    weights[13] = 4;
    weights[14] = 8;
    weights[15] = 16;
    weights[16] = 8;
    weights[17] = 16;

//WEIGHTS SET

    ebmpBYTE *A_h, *B_h;
    ebmpBYTE *A_d, *B_d;
    A_h = &Background.Pixels[0][0].Blue;
    B_h = &Output.Pixels[0][0].Blue;
    dim3 dim_grid, dim_block;

    hipMalloc((void**)&weights_d, sizeof(int)*18 );
    hipMalloc((void**)&A_d, sizeof(ebmpBYTE)*width*height*4);
    hipMalloc((void**)&B_d, sizeof(ebmpBYTE)*width*height*4);

    hipDeviceSynchronize();

    hipMemcpy(weights_d, &weights[0], sizeof(int)*18, hipMemcpyHostToDevice);
    hipMemcpy(A_d, A_h, sizeof(ebmpBYTE)*width*height*4, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    dim3 DimGrid(1, 1, 1);
    dim3 DimBlock(1024, 1, 1);

    gpu_filter<<<DimGrid, DimBlock>>>(A_d, B_d, weights_d, width, height);

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess); //FIXME

    hipMemcpy(B_h, B_d, sizeof(ebmpBYTE)*width*height*4, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    std::string fileout = filename;
    fileout.pop_back();
    fileout.pop_back();
    fileout.pop_back();
    fileout.pop_back();
    string extra = "_cpu_blur.bmp";
    fileout = fileout + extra;
    Output.WriteToFile(fileout.c_str());
    free(A_h);
    free(B_h);

    hipFree(weights_d);
    hipFree(B_d);
    hipFree(A_d);
    return;
}


