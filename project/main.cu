#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.h"
void gpu_blur (string filename)
{
    BMP Background;
    Background.ReadFromFile(filename.c_str());
    int height = Background.TellHeight();
    int width = Background.TellWidth();
    int depth = Background.TellBitDepth();
    BMP Output;
    Output.SetSize( width , height );
    Output.SetBitDepth( 24 );

    hipError_t cuda_ret;

    int *weigths, *weights_d;

    weights = (int*) malloc( sizeof(int)*18 );


//SET WEIGHTS:
    for(int i = 0; i < 9; i++){
        weights[i] = 1;
    }
    weights[10] = 16;
    weights[11] = 8;
    weights[12] = 16;
    weights[13] = 8;
    weights[14] = 4;
    weights[15] = 8;
    weights[16] = 16;
    weights[17] = 8;
    weights[18] = 16;

//WEIGHTS SET

    ebmpBYTE *A_h, *B_h;
    ebmpBYTE *A_d, *B_d;
    A_h = *Background.Pixels;
    B_h = *Output.Pixels;
    dim3 dim_grid, dim_block;

    hipMalloc((void**)&weights_d, sizeof(int)*18 );
    hipMalloc((void**)&A_d, sizeof(ebmpBYTE)*width*height*4);
    hipMalloc((void**)&B_d, sizeof(ebmpBYTE)*width*height*4);

    hipDeviceSynchronize();

    hipMemcpy(weights_d, weights, sizeof(int)*18, hipMemcpyHostToDevice);
    hipMemcpy(A_d, A_h, sizeof(ebmpBYTE)*width*height*4, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    dim3 DimGrid(1, 1, 1);
    dim3 DimBlock(1024, 1, 1);

    gpu_filter<<<DimGrid, DimBlock>>>(A_d, B_d, weights_d, width, height);

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");

    hipMemcpy(B_h, B_d, sizeof(ebmpBYTE)*width*height*4, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    string fileout = filename;
    fileout.pop_back();
    fileout.pop_back();
    fileout.pop_back();
    fileout.pop_back();
    string extra = "_cpu_blur.bmp";
    fileout = fileout + extra;
    Output.WriteToFile(fileout.c_str());
    free(weights);
    free(A_h);
    free(B_h);

    hipFree(weights_d);
    hipFree(B_d);
    hipFree(A_d);
    return(0);
}


