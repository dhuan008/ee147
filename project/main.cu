#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.h"
void gpu_blur (string filename)
{
BMP Background;
Background.ReadFromFile(filename.c_str());
int height = Background.TellHeight();
int width = Background.TellWidth();
int depth = Background.TellBitDepth();
BMP Output;
Output.SetSize( width , height );
Output.SetBitDepth( 24 );

hipError_t cuda_ret;

ebmpBYTE *A_h, *B_h;
ebmpBYTE *A_d, *B_d;
A_h = *Background.Pixels;
B_h = *Output.Pixels;
dim3 dim_grid, dim_block;

hipMalloc((void**)&A_d, sizeof(ebmpBYTE)*width*height);
hipMalloc((void**)&B_d, sizeof(ebmpBYTE)*width*height);

hipDeviceSynchronize();

hipMemcpy(A_d, A_h, sizeof(ebmpBYTE)*width*height, hipMemcpyHostToDevice);

hipDeviceSynchronize();

//FIXME KERNEL CALL HERE

cuda_ret = hipDeviceSynchronize();
if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");

hipMemcpy(B_h, B_d, sizeof(ebmpBYTE)*width*height, hipMemcpyDeviceToHost);

hipDeviceSynchronize();

string fileout = filename;
fileout.pop_back();
fileout.pop_back();
fileout.pop_back();
fileout.pop_back();
string extra = "_cpu_blur.bmp";
fileout = fileout + extra;
Output.WriteToFile(fileout.c_str());

free(A_h);
free(B_h);
hipFree(B_d);
hipFree(A_d);
return(0);
}


