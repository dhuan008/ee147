#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void gpu_filter(ebmpBYTE* pixels, ebmpBYTE* pixels_out, int* weights, int width, int height){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    while (i<(width*height)){
        int column = i % width;
        int row = i / width;
        if (row > 0 && column > 0 && row < height - 1 && column < width - 1){
            int Red = 0;
            int Green = 0;
            int Blue = 0;
            for (int j = column-1; j < column + 2; j++){
                Blue += pixels[((row-1)*width+j)*3+0]*weights[j+1-column]/weights[j+10-column];
                Green += pixels[((row-1)*width+j)*3+1]*weights[j+1-column]/weights[j+10-column];
                Red += pixels[((row-1)*width+j)*3+2]*weights[j+1-column]/weights[j+10-column];

                Blue += pixels[(row*width+j)*3+0]*weights[j+4-column]/weights[j+13-column];
                Green += pixels[(row*width+j)*3+1]*weights[j+4-column]/weights[j+13-column];
                Red += pixels[(row*width+j)*3+2]*weights[j+4-column]/weights[j+13-column];

                Blue += pixels[((row+1)*width+j)*3+0]*weights[j+7-column]/weights[j+16-column];
                Green += pixels[((row+1)*width+j)*3+1]*weights[j+7-column]/weights[j+16-column];
                Red += pixels[((row+1)*width+j)*3+2]*weights[j+7-column]/weights[j+16-column];
            }
	    pixels_out[(row*width+column)*3] = Blue;
	    pixels_out[(row*width+column)*3+1] = Green;
	    pixels_out[(row*width+column)*3+2] = Red;
        }
        i+=1024;
    }
    return;
}
