/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    __shared__ float ds_M[TILE_SIZE][TILE_SIZE];
    __shared__ float ds_N[TILE_SIZE][TILE_SIZE];
    int a1 = blockIdx.x;
    int a2 = blockIdx.y;
    int b1 = threadIdx.x;
    int b2 = threadIdx.y;
    int column = blockDim.x * a1 + b1;
    int row = blockDim.y * a2 + b2;
    float temp = 0;
    int width = m;
    if (n > width) width = n;
    if (k > width) width = k;
    for (int i = 0; i < (width - 1)/TILE_SIZE + 1; i++){ //maybe
	if (row < m && i * TILE_SIZE + b1 < k){
	    ds_M[b2][b1] = A[row * k + i*TILE_SIZE + b1]; //maybe
	}
	else {
	    ds_M[b2][b1] = 0.0;
	}
	if ((i * TILE_SIZE + b2) < k && column < n){	
		ds_N[b2][b1] = B[(i * TILE_SIZE + b2) * n + column]; //maybe
	}
	else {
	    ds_N[b2][b1] = 0.0;
	}
	__syncthreads();
	if (row < m && column < n){ //maybe
	    for (int j = 0; j < TILE_SIZE; j++){
	        temp += ds_M[b2][j] * ds_N[j][b1];
	    }
	}
	__syncthreads();
    }
    if (row < m && column < n){
	C[row * n + column] = temp;
    }
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    //INSERT CODE HERE
    int temp = m;
    if (n > temp) temp = n;
    if (k > temp) temp = k;
    dim3 DimGrid((temp-1)/BLOCK_SIZE+1, (temp-1)/BLOCK_SIZE+1, 1);
    dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
    mysgemm<<<DimGrid, DimBlock>>>(m, n, k, A, B, C);



}


