#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void gpu_filter(ebmpBYTE* pixels, ebmpBYTE* pixels_out, int* weights, int width, int height){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    while (i<(width*height)){
        int column = i % width;
        int row = i / width;
        if (row > 0 && column > 0 && row < height - 1 && column < width - 1){
            int Red = 0;
            int Green = 0;
            int Blue = 0;
            for (int j = column-1; j < column + 2; j++){
                Blue += pixels[((row-1)*width+j)*4+0]*weights[j+1-column]/weights[j+10-column];
                Green += pixels[((row-1)*width+j)*4+1]*weights[j+1-column]/weights[j+10-column];
                Red += pixels[((row-1)*width+j)*4+2]*weights[j+1-column]/weights[j+10-column];

                Blue += pixels[(row*width+j)*4+0]*weights[j+4-column]/weights[j+13-column];
                Green += pixels[(row*width+j)*4+1]*weights[j+4-column]/weights[j+13-column];
                Red += pixels[(row*width+j)*4+2]*weights[j+4-column]/weights[j+13-column];

                Blue += pixels[((row+1)*width+j)*4+0]*weights[j+7-column]/weights[j+16-column];
                Green += pixels[((row+1)*width+j)*4+1]*weights[j+7-column]/weights[j+16-column];
                Red += pixels[((row+1)*width+j)*4+2]*weights[j+7-column]/weights[j+16-column];
            }
        }
        i+=gridDim.x*blockDim.x;
    }
    return;
}
