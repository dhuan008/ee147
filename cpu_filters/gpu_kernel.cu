
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;    
__global__ void gpu_filter(int n, const float *A, const float *B, float* C) {
	unsigned int i = blockIdx.x*256+threadIdx.x;
	if (i < n) C[i] = A[i] + B[i];
	return;
}
